#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

__global__ void vecMax_kernel(double* a, double* b, double* c, unsigned int M) {

    // TODO
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < M) {
        if(a[i] >= b[i])
            c[i] = a[i];
        else
            c[i] = b[i];
    }
}

void vecMax_gpu(double* a, double* b, double* c, unsigned int M) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO
    double *a_d, *b_d, *c_d;
    hipMalloc((void**) &a_d, M*sizeof(double));
    hipMalloc((void**) &b_d, M*sizeof(double));
    hipMalloc((void**) &c_d, M*sizeof(double));


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO
    hipMemcpy(a_d, a, M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, M*sizeof(double), hipMemcpyHostToDevice);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO
    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = M/numThreadsPerBlock;
    vecMax_kernel <<< numBlocks, numThreadsPerBlock >>> (a_d, b_d, c_d, M);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    // TODO
    hipMemcpy(c, c_d, M*sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

